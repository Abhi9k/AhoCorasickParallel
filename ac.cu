#include "hip/hip_runtime.h"
#include "ac.h"
texture<int, hipTextureType1D> tex_state_final;
texture<int, hipTextureType1D> tex_dfa;
texture<int, hipTextureType1D> tex_fail_state;

void
populate_final_states(int* final, int* dfa) {	
	for(int i=0; i<NUM_ROWS; i++) {

		final[i] = dfa[i*NUM_COLS + 0];

	}
}
__global__
void
profanity_filter_cuda(int* dfa, int* fail_state, unsigned char* tweets, bool* valid_state, int offset, int num_tweets, int tweet_length) {

		int num_tweets_per_block = num_tweets/gridDim.x;
		int num_tweets_per_thread = num_tweets/(gridDim.x*blockDim.x);

		int start = blockIdx.x*num_tweets_per_block + threadIdx.x*num_tweets_per_thread;

		int start_ptr = start*tweet_length;


		int curr_state = 0;
		int idx = 0;
		int r_idx = 0;
		unsigned char ch;

		while(r_idx < num_tweets_per_thread && (start + r_idx) < num_tweets) {

			ch = tweets[start_ptr + (r_idx*tweet_length) + idx++];

			if(ch == 10) {
				r_idx += 1;
				curr_state = 0;
				idx = 0;
				continue;
			}
			int ord;
			ord = int(ch) - int('a') + 1;
			if(ch == ' ')
				ord = 28;
			else if(int(ch) == 39)
				ord = 29;

			if(ord <0 && ord >=30)
				continue;

			while(curr_state!=0 && tex1Dfetch (tex_dfa, curr_state*NUM_COLS + ord) == 0){
				curr_state = tex1Dfetch (tex_fail_state, curr_state);
			}

			if(curr_state!=0 || tex1Dfetch (tex_dfa, curr_state*NUM_COLS + ord)!=0) {
				curr_state = tex1Dfetch (tex_dfa, curr_state*NUM_COLS + ord);
				int r = tex1Dfetch ( tex_state_final, curr_state );
				if(r) {
					valid_state[start + r_idx] = true;
					break;
				}

			}

			/* this commented region is our global memory approach */

			// while(curr_state!=0 && dfa[curr_state*NUM_COLS + ord] == 0){
			// 	curr_state = fail_state[curr_state];
			// }

			// if(curr_state!=0 || dfa[curr_state*NUM_COLS + ord]!=0) {
			// 	curr_state = dfa[curr_state*NUM_COLS + ord];
			// 	int r = dfa[curr_state*NUM_COLS] ;
			// 	if(r) {
			// 		valid_state[start + r_idx] = true;
			// 		break;
			// 	}

			// }
		}
}

void
profanity_filter_parallel(int* dfa, int* fail_state, char* tweets, bool* valid_state, int num_tweets, int tweet_length, int num_threads, int num_blocks) {

	if(num_tweets < num_blocks*num_threads) {
		num_blocks = 128;
		num_threads = num_tweets/num_blocks;
	}
	int* d_dfa;
	int* d_fail_state;
	unsigned char* d_tweets;
	bool* d_valid_state;
	int* s_final;
	int* final = (int *) malloc(NUM_ROWS*sizeof(int));
	populate_final_states(final, dfa);

	hipMalloc((void **)&d_fail_state, NUM_ROWS*sizeof(int));
	hipMalloc((void **)&d_valid_state, num_tweets*sizeof(bool));
	hipMalloc((void **)&d_dfa, NUM_COLS*NUM_ROWS*sizeof(int));
	hipMalloc((void **)&s_final, NUM_ROWS*sizeof(int));

	hipMemcpy(d_fail_state, fail_state, NUM_ROWS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dfa, dfa, NUM_ROWS*NUM_COLS*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(s_final, final, NUM_ROWS*sizeof(int), hipMemcpyHostToDevice);

	hipMemset(d_valid_state, false, num_tweets*sizeof(bool));

	hipMalloc((void **)&d_tweets, num_tweets*tweet_length*sizeof(unsigned char));
	hipMemcpy(d_tweets, tweets, num_tweets*tweet_length*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture ( 0, tex_state_final, s_final, NUM_ROWS*sizeof(int) );
	hipBindTexture ( 0, tex_dfa, d_dfa, NUM_ROWS*NUM_COLS*sizeof(int) );
	hipBindTexture ( 0, tex_fail_state, d_fail_state, NUM_ROWS*sizeof(int) );
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(profanity_filter_cuda), hipFuncCachePreferL1);
	profanity_filter_cuda<<<dim3(num_blocks,1,1), dim3(num_threads,1,1)>>>(d_dfa, d_fail_state, d_tweets, d_valid_state, 0, num_tweets, tweet_length);
	hipMemcpy(valid_state, d_valid_state, num_tweets*sizeof(bool), hipMemcpyDeviceToHost);

	hipUnbindTexture ( tex_state_final );
	hipUnbindTexture ( tex_dfa );
	hipUnbindTexture ( tex_fail_state );
	hipFree(d_dfa);
	hipFree(s_final);
	hipFree(d_fail_state);
	hipFree(d_tweets);
	hipFree(d_valid_state);
}
